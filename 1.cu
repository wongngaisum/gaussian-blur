
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

typedef struct {
     unsigned char red,green,blue;
} PPMPixel;

typedef struct {
     int x, y;
     PPMPixel *data;
} PPMImage;

#define CREATOR "COMP3231"
#define RGB_COMPONENT_COLOR 255
#define thread_x 10
#define thread_y 10

#define CUDA_CHECK(err) (cuda_checker(err, __FILE__, __LINE__))

static void cuda_checker(hipError_t err, const char *file, int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

static PPMImage *readPPM(const char *filename)
{
         char buff[16];
         PPMImage *img;
         FILE *fp;
         int c, rgb_comp_color;
         //open PPM file for reading
         fp = fopen(filename, "rb");
         if (!fp) {
              fprintf(stderr, "Unable to open file '%s'\n", filename);
              exit(1);
         }

         //read image format
         if (!fgets(buff, sizeof(buff), fp)) {
              perror(filename);
              exit(1);
         }

    //check the image format
    if (buff[0] != 'P' || buff[1] != '6') {
         fprintf(stderr, "Invalid image format (must be 'P6')\n");
         exit(1);
    }

    //alloc memory form image
    img = (PPMImage *)malloc(sizeof(PPMImage));
    if (!img) {
         fprintf(stderr, "Unable to allocate memory\n");
         exit(1);
    }

    //check for comments
    c = getc(fp);
    while (c == '#') {
    while (getc(fp) != '\n') ;
         c = getc(fp);
    }

    ungetc(c, fp);
    //read image size information
    if (fscanf(fp, "%d %d", &img->x, &img->y) != 2) {
         fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
         exit(1);
    }

    //read rgb component
    if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
         fprintf(stderr, "Invalid rgb component (error loading '%s')\n", filename);
         exit(1);
    }

    //check rgb component depth
    if (rgb_comp_color!= RGB_COMPONENT_COLOR) {
         fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
         exit(1);
    }

    while (fgetc(fp) != '\n') ;
    //memory allocation for pixel data
    img->data = (PPMPixel*)malloc(img->x * img->y * sizeof(PPMPixel));

    if (!img) {
         fprintf(stderr, "Unable to allocate memory\n");
         exit(1);
    }

    //read pixel data from file
    if (fread(img->data, 3 * img->x, img->y, fp) != img->y) {
         fprintf(stderr, "Error loading image '%s'\n", filename);
         exit(1);
    }

    fclose(fp);
    return img;
}

void writePPM(const char *filename, PPMImage *img)
{
    FILE *fp;
    //open file for output
    fp = fopen(filename, "wb");
    if (!fp) {
         fprintf(stderr, "Unable to open file '%s'\n", filename);
         exit(1);
    }

    //write the header file
    //image format
    fprintf(fp, "P6\n");

    //comments
    fprintf(fp, "# Created by %s\n",CREATOR);

    //image size
    fprintf(fp, "%d %d\n",img->x,img->y);

    // rgb component depth
    fprintf(fp, "%d\n",RGB_COMPONENT_COLOR);

    // pixel data
    fwrite(img->data, 3 * img->x, img->y, fp);
    fclose(fp);
}

__global__ void blur_kernel(PPMImage *dev_img, PPMPixel *out_data) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = x + gridDim.x * blockDim.x * y;   // index in whole image

    float filter[] = {0.05, 0.1, 0.05, 0.1, 0.4, 0.1, 0.05, 0.1, 0.05}; // localVar filter

    // width and height of image
    int width = dev_img->x;
    int height = dev_img->y;
    __shared__ PPMPixel data[thread_y + 2][thread_x + 2];   // sharedVar pixels of image

    // empty pixel
    PPMPixel empty;
    empty.red = 0;
    empty.green = 0;
    empty.blue = 0;
    int local_idx = threadIdx.x + threadIdx.y * blockDim.x; // index in block

    // read pixels surrounding block into shared memory
    if (local_idx < blockDim.x + 2) {
        int tmp_x = blockIdx.x * blockDim.x - 1 + local_idx;
        int tmp_y = blockIdx.y * blockDim.y - 1;
        if (tmp_x < 0 || tmp_y < 0 || tmp_x >= blockDim.x * gridDim.x || tmp_y >= blockDim.y * gridDim.y)
            data [0][local_idx] = empty;
        else
            data [0][local_idx] = dev_img->data[tmp_x + tmp_y * gridDim.x * blockDim.x];

        tmp_y = (blockIdx.y + 1) * blockDim.y;   
        if (tmp_x < 0 || tmp_y < 0 || tmp_x >= blockDim.x * gridDim.x || tmp_y >= blockDim.y * gridDim.y)
            data[blockDim.y + 1][local_idx] = empty;
        else
            data[blockDim.y + 1][local_idx] = dev_img->data[tmp_x + tmp_y * gridDim.x * blockDim.x];    
    }

    if (local_idx < blockDim.y) {
        int tmp_x = blockIdx.x * blockDim.x - 1;
        int tmp_y = blockIdx.y * blockDim.y + local_idx;
        if (tmp_x < 0 || tmp_y < 0 || tmp_x >= blockDim.x * gridDim.x || tmp_y >= blockDim.y * gridDim.y)
            data[local_idx + 1][0] = empty;
        else
            data[local_idx + 1][0] = dev_img->data[tmp_x + tmp_y * gridDim.x * blockDim.x];

        tmp_x = (blockIdx.x + 1) * blockDim.x;   
        if (tmp_x < 0 || tmp_y < 0 || tmp_x >= blockDim.x * gridDim.x || tmp_y >= blockDim.y * gridDim.y)
            data[local_idx + 1][blockDim.x + 1] = empty;
        else
            data[local_idx + 1][blockDim.x + 1] = dev_img->data[tmp_x + tmp_y * gridDim.x * blockDim.x];
    }

    // read data into shared memory
    if (idx < width * height)
        data[threadIdx.y + 1][threadIdx.x + 1] = dev_img->data[idx];

    __syncthreads();

    if (idx < width * height) {    // avoid illegal memory access
        out_data[idx].red = data[threadIdx.y][threadIdx.x].red * filter[0] + data[threadIdx.y][threadIdx.x + 1].red * filter[1] + 
                                data[threadIdx.y][threadIdx.x + 2].red * filter[2] + data[threadIdx.y + 1][threadIdx.x].red * filter[3] +
                                data[threadIdx.y + 1][threadIdx.x + 1].red * filter[4] + data[threadIdx.y + 1][threadIdx.x + 2].red * filter[5] +
                                data[threadIdx.y + 2][threadIdx.x].red * filter[6] + data[threadIdx.y + 2][threadIdx.x + 1].red * filter[7] +
                                data[threadIdx.y + 2][threadIdx.x + 2].red * filter[8];

        out_data[idx].green = data[threadIdx.y][threadIdx.x].green * filter[0] + data[threadIdx.y][threadIdx.x + 1].green * filter[1] + 
                                data[threadIdx.y][threadIdx.x + 2].green * filter[2] + data[threadIdx.y + 1][threadIdx.x].green * filter[3] +
                                data[threadIdx.y + 1][threadIdx.x + 1].green * filter[4] + data[threadIdx.y + 1][threadIdx.x + 2].green * filter[5] +
                                data[threadIdx.y + 2][threadIdx.x].green * filter[6] + data[threadIdx.y + 2][threadIdx.x + 1].green * filter[7] +
                                data[threadIdx.y + 2][threadIdx.x + 2].green * filter[8];

        out_data[idx].blue = data[threadIdx.y][threadIdx.x].blue * filter[0] + data[threadIdx.y][threadIdx.x + 1].blue * filter[1] + 
                                data[threadIdx.y][threadIdx.x + 2].blue * filter[2] + data[threadIdx.y + 1][threadIdx.x].blue * filter[3] +
                                data[threadIdx.y + 1][threadIdx.x + 1].blue * filter[4] + data[threadIdx.y + 1][threadIdx.x + 2].blue * filter[5] +
                                data[threadIdx.y + 2][threadIdx.x].blue * filter[6] + data[threadIdx.y + 2][threadIdx.x + 1].blue * filter[7] +
                                data[threadIdx.y + 2][threadIdx.x + 2].blue * filter[8];
    }
}

void your_gaussian_blur_func(PPMImage *img) {

    PPMImage *host_img; // for assigning PPMPixel pointer on device

    host_img = (PPMImage *) malloc(sizeof(PPMImage));
    memcpy(host_img, img, sizeof(PPMImage));

    CUDA_CHECK(hipMalloc((void**)&(host_img->data), img->x * img->y * sizeof(PPMPixel)));  // allocate PPMPixel pointer on device
    CUDA_CHECK(hipMemcpy(host_img->data, img->data, img->x * img->y * sizeof(PPMPixel), hipMemcpyHostToDevice));  // copy PPMPixel data to device

    // PPMPixel data is now on the gpu, now copy the "meta" data to gpu

    PPMImage *dev_img;  // for assigning PPMImage on device
    CUDA_CHECK(hipMalloc((void**)&dev_img, sizeof(PPMImage)));  // allocate memory on device
    CUDA_CHECK(hipMemcpy(dev_img, host_img, sizeof(PPMImage), hipMemcpyHostToDevice));  // copy memory to device

    PPMPixel *out_data;
    CUDA_CHECK(hipMalloc((void**)&(out_data), img->x * img->y * sizeof(PPMPixel)));  // allocate PPMPixel pointer on device

    dim3 threadsPerBlock = dim3(thread_x, thread_y);
    dim3 blocksPerGrid = dim3((img->x + thread_x - 1) / thread_x, (img->y + thread_y - 1) / thread_y);
    
    blur_kernel<<<blocksPerGrid, threadsPerBlock>>>(dev_img, out_data);

    CUDA_CHECK(hipMemcpy(img->data, out_data, img->x * img->y * sizeof(PPMPixel), hipMemcpyDeviceToHost));  // copy memory to host

    CUDA_CHECK(hipFree(out_data));
    CUDA_CHECK(hipFree(host_img->data));
    CUDA_CHECK(hipFree(dev_img));
    free(host_img);
}

int main(){
    // read
    PPMImage *image;
    image = readPPM("input.ppm");

    // record execution time
    float time;
    hipEvent_t start, stop;

    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    CUDA_CHECK(hipEventRecord(start, 0));

    your_gaussian_blur_func(image);

    CUDA_CHECK(hipEventRecord(stop, 0));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&time, start, stop));

    printf("Time to generate:  %3.1f ms \n", time);   

    // write
    writePPM("output.ppm",image);
}